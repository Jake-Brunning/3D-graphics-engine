#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h";
#include ""
#include <iostream>;

#include "Vector.h"
#include "List.h"
#include "LinkFile.cuh"


//moving the camera
__global__ void MoveVectors(Vector* d_vectors, double changeInXYZ, char axis, int N) 
{
	int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (i < N) {
		switch (axis) {
		case 'x':
			d_vectors[i].x += changeInXYZ;
			break;

		case 'y':
			d_vectors[i].y += changeInXYZ;
			break;

		case 'z':
			d_vectors[i].z += changeInXYZ;
			break;
		}
	}
}

__host__ Vector* setUpMoveVectors(double changeInXYZ, char axis, Vector* vectors, int N) {

	Vector* d_vectors;
	hipMalloc(&d_vectors, sizeof(Vector) * N);
	hipMemcpy(d_vectors, vectors, sizeof(Vector) * N, hipMemcpyHostToDevice);

	const int numberOfThreads = 32;
	const int numberOfBlocks = (N / 32) + 1;
	
	MoveVectors << <numberOfBlocks, numberOfThreads >> > (d_vectors, changeInXYZ, axis, N);

	Vector* h_vectors = new Vector[N];
	hipMemcpy(h_vectors, d_vectors, sizeof(Vector) * N, hipMemcpyDeviceToHost);

	return h_vectors;
}