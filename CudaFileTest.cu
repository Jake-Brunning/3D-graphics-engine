#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h";
#include ""
#include "CudaLinkTest.cuh";
#include <iostream>;



__global__ void parrallelExecutionTest(int* d_A, int* d_B, int* d_C) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	d_C[i] = d_A[i] + d_B[i];
}

__host__ void cpuEnterFunction() {
	int N = 100;
	int* h_A = new int[N];
	int* h_B = new int[N];
	int* h_C = new int[N];

	for (int i = 0; i < N; i++) {
		h_A[i] = i;
		h_B[i] = i;
	}

	//initilises the cuda environment
	hipFree(0);

	int* d_A;
	hipMalloc(&d_A, sizeof(int) * N);
	int* d_B;
	hipMalloc(&d_B, sizeof(int) * N);
	int* d_C;
	hipMalloc(&d_C, sizeof(int) * N);

	hipMemcpy(d_A, h_A, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(int) * N, hipMemcpyHostToDevice);

	int amountOfBlocks = 4;
	parrallelExecutionTest << <amountOfBlocks, N / amountOfBlocks >> > (d_A, d_B, d_C);
	hipMemcpy(h_C, d_C, sizeof(int) * N, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		std::cout << h_C[i] << std::endl;
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}